#include "hip/hip_runtime.h"
#include <reduction.hh>
/*
 * \file reduction_v4.cu
 * A reduction implementation without constraint of gridDim.x = 1.
 * blockDim.x = input.size / 2.
 * Without boundrary check now.
 * Tricks:  
 *  1. Reduce control divergence and improve the memory coalescing by rearrange 
 *     the assignment strategy.
 *  2. Use shared memory to minimize global memory accesses.
 *  3. Enable multi-gird kernel using atomicAdd.
 *  4. Use thread coarsening.
 */

__global__ void reduction_v4(float *input, float *output) {
    extern __shared__ float shared_mem[];
    float *input_s = shared_mem;

    uint32_t segment = COARSE_FACTOR * 2 * blockDim.x * blockIdx.x;
    uint32_t i_g = segment + threadIdx.x;
    float sum = input[i_g];
    for(unsigned int tile = 1; tile < COARSE_FACTOR * 2; tile++) {
        sum += input[i_g + tile * blockDim.x];
    }
    input_s[threadIdx.x] = sum;
    for(uint32_t stride = blockDim.x / 2; stride >= 1; stride /= 2) {
        __syncthreads();
        if(threadIdx.x < stride) {
            input_s[threadIdx.x] += input_s[threadIdx.x + stride];
        }
    }

    if(threadIdx.x == 0) {
        atomicAdd(output, input_s[0]);
    }
}

void reduction_v4_invok(float *input, float *output, uint64_t num) {
    assert(num % 2 == 0);
    dim3 blockDim(num / 2, 1, 1);
    dim3 gridDim(ceil(float(num)/ blockDim.x), 1, 1);

    reduction_v4<<<gridDim, blockDim>>>(input, output);
}