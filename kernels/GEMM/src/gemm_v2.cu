#include "hip/hip_runtime.h"
/*
 * gemmv2.cu
 * The Matrix M and N don't have to be square matrices.
 * The size of the shared memory is not determined until runtime.
*/

#include"gemm.hh"


__global__ void matrixMul2(float *M, float *N, float *P, int r, int s, int t, unsigned int Mds_sz, unsigned int Nds_sz){
    
    extern __shared__ char Mds_Nds[];

    float *Mds = (float *) Mds_Nds;
    float *Nds = (float *) (Mds_Nds + Mds_sz);

    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int Row = by * blockDim.y + ty;
    int Col = bx * blockDim.x + tx;

    float Pvalue = 0;
    for(int phase = 0; phase < ceil(s/(float)blockDim.x); phase ++){
        if(Row < r && (phase * blockDim.x + tx) < s)
            Mds[ty * blockDim.x + tx] = M[Row * s + phase * blockDim.x + tx];
        else
            Mds[ty * blockDim.x + tx] = 0.0f;
        
        if(Col < t && (phase * blockDim.x + ty) < s)
            Nds[ty * blockDim.x + tx] = N[(phase * blockDim.x + ty) * t + Col];
        else
            Nds[ty * blockDim.x + tx] = 0.0f;

        __syncthreads();

        if(Row < r && Col < t){
            for(int k = 0; k < blockDim.x; k++){
                Pvalue += Mds[ty * blockDim.x + k] * Nds[k * blockDim.x + tx];
            }
        }
        __syncthreads();
    }

    if(Row < r && Col < t)
        P[Row * t + Col] = Pvalue;
}