#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <string>
#include "histogram.hh"

int main(int argc, char **argv){
    if(argc != 2){
        std::cerr << "Usage: " << argv[0] << "(v0|v1|v2)" << std::endl;
        return -1;
    }
    std::string mode = argv[1];

    int length;
    unsigned int histo[BUCKET];

    FILE *fi;

    fi = fopen("data/input.dat", "rb");
    fread(&length, 1, sizeof(int), fi);
    std::cout << "The length of the random letters: " << length << std::endl;

    char *in = (char *)malloc(length * sizeof(char));

    fread(in, sizeof(char), length, fi);
    fclose(fi);

    if(mode == "v0"){
        std::cout << "The mode is v0" << std::endl;

        auto t1 = std::chrono::steady_clock::now();
        char *in_d;
        unsigned int *histo_d;

        hipMalloc(&in_d, length * sizeof(char));
        hipMalloc(&histo_d, BUCKET * sizeof(unsigned int));

        hipMemcpy(in_d, in, length * sizeof(char), hipMemcpyHostToDevice);

        dim3 blockDim(1024, 1, 1);
        dim3 gridDim(ceil(float(length) / blockDim.x), 1, 1);

        histogram_v0<<<blockDim, gridDim>>>(in_d, length, histo_d);
        hipDeviceSynchronize();

        hipMemcpy(histo, histo_d, BUCKET * sizeof(unsigned int), hipMemcpyDeviceToHost);

        auto t2 = std::chrono::steady_clock::now();
        int d1 = std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count();

        std::cout << d1 << std::endl;

        for(int i = 0; i < BUCKET; i++){
            std::cout << histo[i] << std::endl;
        } 

        free(in);
        hipFree(in_d); hipFree(histo_d);
    }
    else if(mode == "v1") {
        std::cout << "The mode is v1" << std::endl;

        auto t1 = std::chrono::steady_clock::now();
        char *in_d;
        unsigned int *histo_d;

        hipMalloc(&in_d, length * sizeof(char));
        hipMalloc(&histo_d, BUCKET * sizeof(unsigned int));

        hipMemcpy(in_d, in, length * sizeof(char), hipMemcpyHostToDevice);

        dim3 blockDim(1024, 1, 1);
        dim3 gridDim(ceil(float(length) / blockDim.x), 1, 1);

        histogram_v1<<<blockDim, gridDim>>>(in_d, length, histo_d);
        hipMemcpy(histo, histo_d, BUCKET * sizeof(unsigned int), hipMemcpyDeviceToHost);

        auto t2 = std::chrono::steady_clock::now();

        int d1 = std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count();

        std::cout << d1 << std::endl;

        for(int i = 0; i < BUCKET; i++){
            std::cout << histo[i] << std::endl;
        }
        free(in);
        hipFree(in_d); hipFree(histo_d);
    }
    else{
        std::cout << "The mode is illegal" << std::endl;
        free(in);
        return -1;
    }

    return 0;
}